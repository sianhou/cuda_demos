#include "hip/hip_runtime.h"
#include "test.cuh"
#include "fstream"
#include "iomanip"

#define smA(i, j) shared_a[j][i]
#define smB(i, j) shared_b[j][i]

template<int BLK>
__global__ void sgemm(const float *a, const float *b, float *c, int M, int N, int K) {
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int lty = threadIdx.y;
    int ltx = threadIdx.x;
    int by = blockIdx.y;
    int bx = blockIdx.x;

    __shared__ float shared_a[BLK][BLK];
    __shared__ float shared_b[BLK][BLK];

    const float *ptr_a = a + bx * BLK;
    const float *ptr_b = b + by * BLK * K;
    float sum = 0.f;

    for (int kk = 0; kk < K; kk += BLK) {
        smA(ltx, lty) = ptr_a[ltx + lty * M];
        smB(ltx, lty) = ptr_b[ltx + lty * K];
        __syncthreads();

#pragma unroll
        for (int i = 0; i < BLK; ++i) {
            sum += smA(ltx, i) * smB(i, lty);
        }
        __syncthreads();

        ptr_a += BLK * M;
        ptr_b += BLK;
    }
    c[tx + ty * M] = sum;
}

template<int BLK>
Result test_cugemm(int size, int blk, int niter) {

    int M = size, N = size, K = size;
    dim3 grid, block;
    Result res;
    float sum_of_time, sum_of_gfops;
    res.size = size;

    Test test(sgemm<BLK>, M, N, K);

    block.y = blk;
    block.x = blk;

    grid.y = (M + block.y - 1) / block.y;
    grid.x = (N + block.x - 1) / block.x;

    std::cout << "M = N = K = " << size << std::endl;
    std::cout << "grid.z x grid.y x grid.x = " << grid.z << " x " << grid.y << " x " << grid.x << std::endl;
    std::cout << "block.z x block.y x block.x = " << block.z << " x " << block.y << " x " << block.x << std::endl;

    // warm up and check out
    test.CheckResult(grid, block);

    // cublas
    test.RunCublas(niter);
    std::cout << std::endl << "cublas:" << std::endl;
    sum_of_time = 0;
    sum_of_gfops = 0;
    for (int i = 0; i < niter; ++i) {
        std::cout << i << ": runtime = " << test.watch[i] << ", gflops = " << test.gflops[i] << std::endl;
        sum_of_time += test.watch[i];
        sum_of_gfops += test.gflops[i];
    }
    res.elapsed_cublas = sum_of_time / niter;
    res.gflops_cublas = sum_of_gfops / niter;

    // sgemm
    test.RunSgemm(grid, block, niter);
    std::cout << std::endl << "sgemm:" << std::endl;
    sum_of_time = 0;
    sum_of_gfops = 0;
    for (int i = 0; i < niter; ++i) {
        std::cout << i << ": runtime = " << test.watch[i] << ", gflops = " << test.gflops[i] << std::endl;
        sum_of_time += test.watch[i];
        sum_of_gfops += test.gflops[i];
    }
    res.elapsed_sgemm = sum_of_time / niter;
    res.gflops_sgemm = sum_of_gfops / niter;

    return res;
}

int main() {
    Result res;
    std::ofstream ofs("sgemm_v2.txt");

    for (int s = 1024; s <= 1024; s += 32) {
        res = test_cugemm<16>(s, 16, 10);

        ofs << std::setw(4) << res.size << " ";
        ofs << std::setiosflags(std::ios::fixed) << std::setprecision(2);
        ofs << std::setw(8) << res.elapsed_cublas << " ";
        ofs << std::setw(8) << res.gflops_cublas << " ";
        ofs << std::setw(8) << res.elapsed_sgemm << " ";
        ofs << std::setw(8) << res.gflops_sgemm << std::endl;
    }

    ofs.close();
}

